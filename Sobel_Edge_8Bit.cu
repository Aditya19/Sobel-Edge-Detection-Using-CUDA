#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<xtiffio.h>
#include<geotiff.h>
#include<tiffio.h>
#include<math_functions.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>

void edge_gpu(unsigned char *buff , unsigned char *buffer_out , int w , int h);
void checkCUDAError(const char* msg);

__global__ void edge_gpu(unsigned char* buff , unsigned char* buffer_out , int w , int h)
{
  int x = blockIdx.x * blockDim.x +threadIdx.x ;
	int y = blockIdx.y * blockDim.y +threadIdx.y; 
	int width = w , height = h ;
	
	if((x>=0 && x < width) && (y>=0 && y<height))
	{
		int hx = -buff[width*(y-1) + (x-1)] + buff[width*(y-1)+(x+1)]
			 -2*buff[width*(y)+(x-1)] + 2*buff[width*(y)+(x+1)]
			 -buff[width*(y+1)+(x-1)] + buff[width*(y+1)+(x+1)];

		int vx = buff[width*(y-1)+(x-1)] +2*buff[width*(y-1)+(x+1)] +buff[width*(y-1)+(x+1)]
			 -buff[width*(y+1)+(x-1)] - 2*buff[width*(y+1)+(x)] - buff[width*(y+1)+(x+1)];
 
		hx = hx/5;
		vx = vx/5;

		int val = (int)sqrt((float)(hx) * (float)(hx) + (float)(vx) * (float)(vx));					

		buffer_out[y * width + x] = (unsigned char) val;							
	}
}
//=========	End Of GPU Function	=========

void checkCUDAError(const char* msg) 
{
	hipError_t err = hipGetLastError();
  	if (hipSuccess != err) 
  	{
    		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    		exit(EXIT_FAILURE);
  	}
}

//=========  Main function  =========

int main(int argc , char** argv)
{
	TIFF *input;
	uint16 photo, bps, spp, rps, comp, pconfig;
	uint32 width, height;
	tsize_t strip_size;
	int strip_max;
	unsigned long buffer_size, offset = 0;
	long int result;
	unsigned char *buffer,*buffer_dev,*buffer_out;
	struct timeval gpu_t1,gpu_t2,gpu_tot;
	
	char infile[100]="parzen.tif", outfile[20]="edge.tif";

	printf("\nSimpe Image Edge Detection Demo Using CUDA.");
	printf("\nEnter .tif Image (8-Bit) : ");
	scanf("%s",infile);

	if((input = XTIFFOpen(infile, "r")) == NULL)
	{
		printf("\nCan not open image %s.", infile);
		exit(42);
	}
	else
	{
		printf("\nTIFF Image Opened Successfully.");
	}

//==========  Extract Image properties....!!  ==========

	TIFFGetField(input, TIFFTAG_BITSPERSAMPLE, &bps);
        TIFFGetField(input, TIFFTAG_SAMPLESPERPIXEL, &spp);
        TIFFGetField(input, TIFFTAG_IMAGEWIDTH, &width);   
        TIFFGetField(input, TIFFTAG_IMAGELENGTH, &height); 
        TIFFGetField(input, TIFFTAG_ROWSPERSTRIP, &rps);   
        TIFFGetField(input, TIFFTAG_COMPRESSION,&comp);    
        TIFFGetField(input, TIFFTAG_PHOTOMETRIC,&photo);   
        TIFFGetField(input, TIFFTAG_PLANARCONFIG, &pconfig);

	printf("\nImage Properties Are : ");

	printf("\nImage Width : %d, Image Height : %d .", width, height);
	printf("\nPhotometric : %d.", photo);		
	printf("\nBits Per Pixels : %d.",bps);		
	printf("\nSamples Per Pixel : %d.",spp);	
	printf("\nRows Per Strip : %d.",rps);
	printf("\nCompression : %d.",comp);
	printf("\nPlanerconfig : %d.",pconfig);

//==========

	strip_size = TIFFStripSize(input);
	strip_max = TIFFNumberOfStrips(input);

	buffer_size = strip_max * strip_size;

	printf("\nNumber of Strips : %d.", strip_max);
	printf("\nStrip Size : %d.", strip_size);
	printf("\nNo of pixels : %ld.", buffer_size);
	
	buffer = (unsigned char *)malloc(buffer_size);	
	
	if(!buffer)
	{
		fprintf(stderr, "\nCould not allocate buffer for uncompressed Image.");
		exit(42);
	}	

	printf("\nNumber of Bytes Required : %d",buffer_size);
	offset = 0;
	for(int i=0;i<strip_max;i++)
	{
		if((result = (long int)TIFFReadEncodedStrip(input, i, buffer + offset, strip_size)) == -1)
		{
			fprintf(stderr,"\nReading Error in Input Strip No : %d");
			exit(42);
		}
		offset = offset + result;
	}

	//	CUDA-GPU Code for Edge Detection .....

	hipMalloc((void**)&buffer_out,buffer_size);
	checkCUDAError("Memory Allocation");
	
	hipMalloc((void**)&buffer_dev,buffer_size);
	checkCUDAError("Memory Allocation");
	
	dim3 threadsPerBlock(8,8);
	dim3 numBlocks((width)/8,(height)/8);

//==========  create a stream  ==========

	hipStream_t stream;
  	hipStreamCreate(&stream);
  	
	gettimeofday(&gpu_t1,NULL);
	
	hipMemcpy(buffer_dev , buffer , buffer_size , hipMemcpyHostToDevice);
	checkCUDAError("Memory Copy From Host To Device");
	
	edge_gpu<<< numBlocks , threadsPerBlock , 0 , stream >>>(buffer_dev , buffer_out, width , height);
	checkCUDAError("Kernel");
		
	unsigned char* buf = (unsigned char*) malloc(buffer_size);
	if(!buf)
	{
		fprintf(stderr,"\nCould no allocate buffer...Insufficient Memory.");
	}

	hipMemcpy(buf , buffer_out , buffer_size  , hipMemcpyDeviceToHost);
	checkCUDAError("Memory Copy From Device To Host");

	gettimeofday(&gpu_t2,NULL);
	timersub(&gpu_t2,&gpu_t1,&gpu_tot);

        XTIFFClose(input);

	if((input = XTIFFOpen(outfile, "w")) == NULL)
	{
		printf("\nCan not open output file.");
		exit(42);
	}

	// Write Image properties into output file....

        TIFFSetField(input, TIFFTAG_IMAGEWIDTH, width);
        TIFFSetField(input, TIFFTAG_IMAGELENGTH, height);
        TIFFSetField(input, TIFFTAG_BITSPERSAMPLE, bps);
        TIFFSetField(input, TIFFTAG_SAMPLESPERPIXEL, 1);
        TIFFSetField(input, TIFFTAG_ROWSPERSTRIP, height);
        TIFFSetField(input, TIFFTAG_COMPRESSION, comp);
        TIFFSetField(input, TIFFTAG_PHOTOMETRIC, photo);
        TIFFSetField(input, TIFFTAG_PLANARCONFIG, pconfig);	

	TIFFWriteEncodedStrip(input,0, buf, buffer_size);
	
	printf("\n\nTime Required for GPU : ");
	printf(" %d Seconds , %d Milliseconds.",gpu_tot.tv_sec,gpu_tot.tv_usec);

        XTIFFClose(input);

	hipFree(buffer_dev);
	hipFree(buffer_out);
	hipStreamDestroy(stream);	

	free(buf);
	
	printf("\n\nDONE...!!...Copy Successful...!!\n\n");

	return 1;
}

